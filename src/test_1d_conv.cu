#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <cstdio>
#include <cstdlib>
#include <math.h>

#include "cuPrintf.cu"
#include "cuPrintf.cuh"

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans)                           \
	{                                                 \
		gpuAssert((ans), (char *)__FILE__, __LINE__); \
	}
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
				hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

#ifdef __APPLE__
/* Shim for Mac OS X (use at your own risk ;-) */
#include "apple_pthread_barrier.h"
#endif /* __APPLE__ */

#define CPNS 2.9 /* Cycles per nanosecond -- Adjust to your computer, for example a 3.2 GhZ GPU, this would be 3.2 */

// Things for 1D_Conv
#define PRINT_TIME	1	// Whether we want to measure time cost (1/0)
#define P_ARR_LEN 	1024	// array/vector size for output (P), must be multiple of NUM_THREADS_PER_BLOCK ==> Otherwise, you will have unmatched result...
#define MASK_WIDTH	3	// array size for mask (M)

// Things for running on GPU
#define NUM_THREADS_PER_BLOCK 16 // Number of threads per block
#define TOL 0.05

#define ITERATIONS 2000
#define MINVAL 0.0
#define MAXVAL 10.0

// dim3 dimGrid( ceil(SM_ARR_LEN/NUM_THREADS_PER_BLOCK), ceil(SM_ARR_LEN/NUM_THREADS_PER_BLOCK));   // Shape of grid
// dim3 dimBlock(NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK);    // Each block has shape of <16, 16>, so 256 threads/block

/* Prototypes */
void initializeArray1D(float *arr, int len, int HALO_CELL, int seed);
void print_1D_array(float *arr, int arrlen);
void print_2D_array(float *arr, int arrlen);
void conv_1D(float *N, float *M, float *P, int mask_width, int N_rowlen);
__global__ void cuda_conv_1D_single_block(float *N, float *M, float *P, int mask_width, int N_rowlen);
__global__ void cuda_conv_1D_multi_block(float *N, float *M, float *P, int mask_width, int N_rowlen);
__global__ void cuda_conv_1D_multi_block_with_constant_mask(float *N, float *P, int mask_width, int N_rowlen);
__global__ void cuda_conv_1D_tiled_kernel(float *N, float *P, int mask_width, int N_rowlen);

/* Things to put into device constant memory */
__constant__ float d_mask_constant[MASK_WIDTH];


/* -=-=-=-=- Time measurement by clock_gettime() -=-=-=-=- */
/*
  As described in the clock_gettime manpage (type "man clock_gettime" at the
  shell prompt), a "timespec" is a structure that looks like this:

		struct timespec {
		  time_t   tv_sec;   // seconds
		  long     tv_nsec;  // and nanoseconds
		};
 */

double interval(struct timespec start, struct timespec end)
{
	struct timespec temp;
	temp.tv_sec = end.tv_sec - start.tv_sec;
	temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	if (temp.tv_nsec < 0)
	{
		temp.tv_sec = temp.tv_sec - 1;
		temp.tv_nsec = temp.tv_nsec + 1000000000;
	}
	return (((double)temp.tv_sec) + ((double)temp.tv_nsec) * 1.0e-9);
}
/*
	 This method does not require adjusting a #define constant

  How to use this method:

	  struct timespec time_start, time_stop;
	  clock_gettime(CLOCK_REALTIME, &time_start);
	  // DO SOMETHING THAT TAKES TIME
	  clock_gettime(CLOCK_REALTIME, &time_stop);
	  measurement = interval(time_start, time_stop);
 */

/* -=-=-=-=- End of time measurement declarations =-=-=-=- */
// nvcc -g -G -lrt -lm  src/test_1d_conv.cu -o test_1d_conv
/*****************************************************************************/
int main(int argc, char *argv[])
{
	int i;
	int start_point = 0;
	// 1D_Conv variables
	int HALO_CELL	= ceil(MASK_WIDTH / 2);	/* 2 extra rows/columns for "ghost zone". */
	int N_ARR_LEN 	= P_ARR_LEN+HALO_CELL*2;	// 1 for start, 1 for end	
	// GPU Timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	struct timespec time_start, time_stop;
	double elapsed_cpu;
	printf("\n\n");
	printf("Size of HALO_CELL: %d\n", HALO_CELL);
	printf("Length of input array(N): %d\n", N_ARR_LEN);
	printf("Length of mask array(M): %d\n", MASK_WIDTH);
	printf("Length of output array(P): %d\n", P_ARR_LEN);

	/* ======================Memory Allocation (CPU & GPU) =============== */
	// size_t alloc_size = (N_ARR_LEN) * sizeof(float);
	// printf("%d element bing allocated, and %ld size in byte\n", N_ARR_LEN, alloc_size);
	
	// Allocate arrays on host memory (calloc will use zero-initialization)
	float *h_input 			= (float *)calloc(N_ARR_LEN, sizeof(float));
	float *h_mask 			= (float *)calloc(MASK_WIDTH, sizeof(float));
	float *h_output_gold 	= (float *)calloc(P_ARR_LEN, sizeof(float));	// result computed in CPU
	float *h_output_data 	= (float *)calloc(P_ARR_LEN, sizeof(float));	// result computed in GPU

	// Allocate GPU memory
	float *d_input, *d_mask, *d_output_data; // Arrays on GPU global memory
	hipMalloc((void **)&d_input, 		N_ARR_LEN*sizeof(float));
	hipMalloc((void **)&d_mask, 		MASK_WIDTH*sizeof(float));
	hipMalloc((void **)&d_output_data, P_ARR_LEN*sizeof(float));


	/* ====================== Memory Initialization =============== */
	// Intialize arrays on host memory
	printf("\nInitializing the arrays ...\n");
	// Arrays are initialized with a known seed for reproducability
	initializeArray1D(h_input, N_ARR_LEN, HALO_CELL, 2453);
	printf("\t... done\n\n");

	// Intialize value for h_input:
	for (i = 0; i < N_ARR_LEN-1; i++){
		h_input[i] = (float)i;
		if (i < 5) printf("h_input[%d] = %.2f\n", i, h_input[i]);
	}
	printf("...\n");
	
	// Intialize value for h_mask
	// h_input 			= {0, 1, 2, 3, 4, 5, 0};
	h_mask[0] = 0.3; h_mask[1] = 0.2; h_mask[2] = 0.8;
	for (i=0; i<MASK_WIDTH; i++){
		printf("h_mask[%d] = %.2f\n", i, h_mask[i]);
	}
	
	// Verify initialized value
	// print_1D_array(h_output_gold, P_ARR_LEN);
	// print_1D_array(h_output_data, P_ARR_LEN);
	// print_1D_array(h_input, N_ARR_LEN);
	// print_1D_array(h_mask, MASK_WIDTH);


	/* ====================== Running code on CPU =============== */
	printf("Running code in CPU \n");
	clock_gettime(CLOCK_REALTIME, &time_start);
	conv_1D(h_input, h_mask, h_output_gold, MASK_WIDTH, N_ARR_LEN);
	clock_gettime(CLOCK_REALTIME, &time_stop);
	elapsed_cpu = interval(time_start, time_stop);
	printf("Finished running 1D conv in CPU \n");

	printf("All times are in cycles (if CPNS is set correctly in code)\n");
	printf("\n");
	printf("N_lenth, Mask_length, output_length, 1D conv time(msec)\n");
	printf("%7d, \t%12d, \t%13d, \t%13.4g", N_ARR_LEN, MASK_WIDTH, P_ARR_LEN, (double)CPNS * 1.0e3 * elapsed_cpu);
	printf("\n");


	/* ====================== Running code on GPU =============== */
	printf("==========> All CPU tests are done! Now, running GPU code!\n");
	// Select GPU
	CUDA_SAFE_CALL(hipSetDevice(0));
#if PRINT_TIME
	// Create the cuda events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Record event on the default stream
	hipEventRecord(start, 0);
#endif

	/* ====================== Transfer data to GPU/Device =============== */
	// Transfer the arrays to the GPU memory
	CUDA_SAFE_CALL(hipMemcpy(d_input,		h_input,		N_ARR_LEN*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_mask, 		h_mask, 		MASK_WIDTH*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_output_data,h_output_data,	P_ARR_LEN*sizeof(float), hipMemcpyHostToDevice));

	// Transfer M to device constant memory
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_mask_constant), h_mask, MASK_WIDTH*sizeof(float)));	//hipMemcpyToSymbol(HIP_SYMBOL(dest), src, size)
	
	// Launch the kernel
	cudaPrintfInit();
	// dim3 dimGrid(ceil(P_ARR_LEN / 16), ceil(P_ARR_LEN / 16)); // Shape of grid = # of elements in a row divided by the number of threads per block row
	// dim3 dimBlock(16, 16);
	dim3 dimGrid(ceil(P_ARR_LEN / NUM_THREADS_PER_BLOCK), 1); // Shape of grid = # of elements in a row divided by the number of threads per block row
	dim3 dimBlock(NUM_THREADS_PER_BLOCK, 1);
	
	int taskid = 3;		// 1(1D single block) --> 2(1D multi-block) --> 3(1D tiled algo with shared memory)
	printf("==============>Running taskid #: %d on GPU!\n", taskid);
	printf("1(1D single block) --> 2(1D multi-block) --> 3(1D mulit-block with constant mask) --> 4(tiled algo with shared memory)\n");
	switch (taskid) {
		case 1:
			// single block, each thread compute single output
			cuda_conv_1D_single_block<<<dimGrid, dimBlock>>>(d_input, d_mask, d_output_data, MASK_WIDTH, N_ARR_LEN);
			break;
		case 2:
			cuda_conv_1D_multi_block<<<dimGrid, dimBlock>>>(d_input, d_mask, d_output_data, MASK_WIDTH, N_ARR_LEN);
			break;
		case 3:
			cuda_conv_1D_multi_block_with_constant_mask<<<dimGrid, dimBlock>>>(d_input, d_output_data, MASK_WIDTH, N_ARR_LEN);
			break;
		case 4:
			cuda_conv_1D_tiled_kernel<<<dimGrid, dimBlock>>>(d_input, d_output_data, MASK_WIDTH, N_ARR_LEN);
			break;
		default:
			printf("ERROR: You hit an error, no such taskid # %d!n", taskid);
			break;
	}
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();

	// Check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());

	// Transfer the results back to the host
	CUDA_SAFE_CALL(hipMemcpy(h_input, 		d_input, 		N_ARR_LEN*sizeof(float),  hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(h_mask,  		d_mask, 		MASK_WIDTH*sizeof(float), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(h_output_data,d_output_data,	P_ARR_LEN*sizeof(float), hipMemcpyDeviceToHost));

#if PRINT_TIME
	// Stop and destroy the timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	printf("\nGPU time: %f (msec)\n", elapsed_gpu);
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif

	// Comparing the result we obtained from kernel function and regular SOR:
	int errCount = 0;
	int zeroCount = 0;
	float *relative_error = (float *)malloc( P_ARR_LEN * sizeof(float));
	for (i = 0; i < P_ARR_LEN; i++){
		relative_error[i] = abs(h_output_gold[i] - h_output_data[i]) / h_output_gold[i] * 100;
		if (relative_error[i] > TOL){
			printf("FAILED in %d:\t%.4f\t%.4f\t%.2f %%\n", i, h_output_gold[i], h_output_data[i], relative_error[i]);
			errCount++;
		}
		if (h_output_data[i] == 0.0){
			zeroCount++;
		}
	}

	// double error_rate = errCount/(arrLen*arrLen) * 100;
	if (errCount > 0){
		printf("\n@ERROR: TEST FAILED: %d/%d results did not match\n", errCount, P_ARR_LEN);
	}else if (zeroCount > 0){
		printf("\n@ERROR: TEST FAILED: %d/%d results (from GPU) are zero\n", zeroCount, P_ARR_LEN);
	}else{
		printf("\nTEST PASSED: All results matched\n");
	}

	printf("\n");
	start_point = 0;
	int end_point = P_ARR_LEN < 50 ? P_ARR_LEN : 50;
	for (i = start_point; i < end_point; i++){
		printf("%d:\t%.4f\t%.4f\n", i, h_output_gold[i], h_output_data[i]);
	}

	// Free-up device and host memory
	CUDA_SAFE_CALL(hipFree(d_input));
	CUDA_SAFE_CALL(hipFree(d_mask));
	CUDA_SAFE_CALL(hipFree(d_output_data));

	// free(h_input);
	// free(h_mask);
	// free(h_output_data);
	// free(h_output_gold);

	return 0;
} /* end main */


/************************ Some helper function  ******/
void initializeArray1D(float *arr, int len, int HALO_CELL, int seed){
	int i;
	float randNum;
	srand(seed);
	if (len > 0){
		for (i = HALO_CELL; i < len-HALO_CELL; i++){
			//Randomly initialize each cell, but leave halo cell to zero
			randNum = (float)rand();
			arr[i] = randNum;
		}
	}
}

/* print all elements of an array */
void print_1D_array(float *arr, int arrlen){
	int i;
	printf("row length = %d\n", arrlen);
	for (i = 0; i < arrlen; i++) {
		printf("%.4f ", arr[i]);
	}
	printf("\n");
}

/* print all elements of an matrix */
void print_2D_matrix(float *arr, int row_len){
  int i, j;
  printf("row length = %d\n", row_len);
  for (i = 0; i < row_len; i++) {
    for (j = 0; j < row_len; j++)
      printf("%.4f ",arr[i*row_len+j]);
    printf("\n");
  }
}


/************************************/

/* matrix-array multiplication */
void conv_1D(float *N, float *M, float *P, int mask_width, int N_rowlen){
	// matrix a of size 1 x n (array)
	// matrix b of size n x p
	// matrix result of size 1 x p (array)
	// result = a * b
	int i;
	float Pvalue;
	for (i = 0; i < N_rowlen; i++){
		Pvalue = 0;
		for (int j = 0; j < mask_width; j++){
			Pvalue += N[i + j] * M[j];
		}
		P[i] = Pvalue;
	}
}

/************ CPU implementation
 * The idea for this implementation, is to chopping off the entire matrix by block/tile, so that each thread is responsible for a single block.
 */
__global__ void cuda_conv_1D_single_block(float *N, float *M, float *P, int mask_width, int N_rowlen){
	/*
	Input parameter:
		float *N: pointer to input array N
		float *M: pointer to input mask M
		float *P: pointer to output array P
		int Mask_Width: size of mask, e.g., (1, len(M))
		int Width: size of input and output array Width, e.g., (1, n=len(N))
	*/
	int i = threadIdx.x;	// i is [0, P_ARR_LEN-1]
	float Pvalue = 0;
	int j;

	// kernel body
	// int halo_cell = ceil(mask_width /2);	// halo_cell = 1
	// int N_start_point = i - halo_cell;		// range bet [-1, P_ARR_LEN-2]
	for (j = 0; j < mask_width; j++) {
		Pvalue += N[i + j]*M[j];
	}
	P[i] = Pvalue;

}

__global__ void cuda_conv_1D_multi_block(float *N, float *M, float *P, int mask_width, int N_rowlen){
	int i = blockIdx.x*blockDim.x + threadIdx.x;	// i is [0, P_ARR_LEN-1]
	float Pvalue = 0;

	// Return directly, if threadIdx exceed the size of P
	if (i>= P_ARR_LEN) return;
	for (int j = 0; j < mask_width; j++) {
		Pvalue += N[i + j]*M[j];
	}
	P[i] = Pvalue;
}

__global__ void cuda_conv_1D_multi_block_with_constant_mask(float *N, float *P, int mask_width, int N_rowlen){
	int i = blockIdx.x*blockDim.x + threadIdx.x;	// i is [0, P_ARR_LEN-1]
	float Pvalue = 0;

	// Return directly, if threadIdx exceed the size of P
	if (i>= P_ARR_LEN) return;
	for (int j = 0; j < mask_width; j++) {
		Pvalue += N[i + j]* d_mask_constant[j];
	}
	P[i] = Pvalue;
}


__global__ void cuda_conv_1D_tiled_kernel(float *N, float *P, int mask_width, int N_rowlen){
	int i = blockIdx.x*blockDim.x + threadIdx.x;	// i is [0, P_ARR_LEN-1]

	


	float Pvalue = 0;
	// Return directly, if threadIdx exceed the size of P
	if (i>= P_ARR_LEN) return;
	for (int j = 0; j < mask_width; j++) {
		Pvalue += N[i + j]* d_mask_constant[j];
	}
	P[i] = Pvalue;
}
